/*******************************************************************************************
 * Filename: main.cu
 * Author: Drew Hans (github.com/drewhans555)
 * Description: This program creates a simple feed-forward artificial neural network
 *              and trains it on the CPU or GPU. The user will input (1) the number
 *              of layers (not including the input layer, which is required), (2) the
 *              number of neurons in each layer (including the input layer), and (3)
 *              whether to run on the CPU or GPU
 *******************************************************************************************
 */

#define MAXINPUT 32
#define DEBUG

#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""

#include "./neuronmancer.h"

#include "./helperfunctions.cu"
#include "./readmodel.cu"
#include "./savemodel.cu"

#include "./loadinput.cu"

#include "./activationfunctions.cu"
#include "./combinationfunctions.cu"
#include "./costfunctions.cu"

#include "./feedforwardfunctions.cu"
#include "./backpropagationfunctions.cu"

#include "./ui_create.cu"
#include "./ui_train.cu"
#include "./ui_evaluate.cu"

/* main - the program starts here */
int main(int argc, char * argv[]) {
    printf("Starting %s...\n", argv[0]);
    printf("Lets create a simple artificial neural network!\n");
    ui_create();

    // declare our host variables
    char inputBuffer[MAXINPUT]; // store the user's input (gets recycled a lot)
/*    int numberOfLayers; // store the total number of layers in the network
    int numberOfNeuronsTotal; // store the total number of neurons in our neural network
    int numberOfWeightsTotal; // store the total number of weights in our neural network
    int* numberOfNeuronsPerLayer; // store the total number of neurons in each layer in our neural network in a 1d array of size numberOfLayers
    int* numberOfWeightsPerLayer; // store the total number of weights between each layer in our neural network in a 1d array of size numberOfLayers-1
    int* firstNeuronIndexPerLayer; // store the indexes of each layer's first neuron value
    int* firstWeightIndexPerLayer; // store the indexes of each layer's first weight value
    double* neurons; // store the neuron values of our neural network in a 1d array of size neuronSize (1d arrays are easy to work with in CUDA)
    double* weights; // store the weight values of our neural network in a 1d array of size weightSize (1d arrays are easy to work with in CUDA)
    double* outputExpected; // store the outputExpected output values for backpropagation
    double* neuronErrors; // store the error "cost" of each neuron during backpropagation
    char runOn; // store the user's choice of host or device
    int epochs = 5; // store the number of epochs for training
    double learningRate = 0.5; // store the rate that our network will learn

    // declare our device variables
    int* deviceNumberOfNeuronsPerLayer;
    int* deviceNumberOfWeightsPerLayer;
    double* deviceNeurons;
    double* deviceWeights;
    double* deviceWeightCosts;

    // declare our cudaStatus variable
    // hipError_t cudaStatus;

    printf("For the following please enter a positive number with no spaces, commas, or decimal points and in less than 31 characters.\n");

    // get the number of layers in the ANN
    printf("How many hidden layers do you want this network to have?\n~");
    fgets(inputBuffer, MAXINPUT, stdin); // read the user's input
    sscanf(inputBuffer, "%d", &numberOfLayers); // format and dump the user's input
    numberOfLayers = numberOfLayers + 2; // account for the input layer

    // dynamically allocate memory for our variables
    numberOfNeuronsPerLayer = (int *) malloc(numberOfLayers * sizeof(int)); //malloc allocates a chunk of host memory
    numberOfWeightsPerLayer = (int *) malloc(numberOfLayers * sizeof(int)); //malloc allocates a chunk of host memory
    firstNeuronIndexPerLayer = (int *) malloc(numberOfLayers * sizeof(int)); //malloc allocates a chunk of host memory
    firstWeightIndexPerLayer = (int *) malloc(numberOfLayers * sizeof(int)); //malloc allocates a chunk of host memory

    // get the number of neurons in input layer in the ANN
    printf("How many neurons do you want the input layer to have?\n~");
    fgets(inputBuffer, MAXINPUT, stdin); // read the user's input
    sscanf(inputBuffer, "%d", &numberOfNeuronsPerLayer[0]); // format and dump the user's input
    for (int i = 1; i < numberOfLayers - 1; i++) {
        printf("How many neurons do you want hidden layer %d to have?\n~", i);
        fgets(inputBuffer, MAXINPUT, stdin); // read the user's input
        sscanf(inputBuffer, "%d", &numberOfNeuronsPerLayer[i]); // format and dump the user's input
    }
    printf("How many neurons do you want the output layer to have?\n~");
    fgets(inputBuffer, MAXINPUT, stdin); // read the user's input
    sscanf(inputBuffer, "%d", &numberOfNeuronsPerLayer[numberOfLayers - 1]); // format and dump the user's input

    printf("Do you want to run on the host CPU or device GPU?\nEnter h for host or d for device:\n~");
    fgets(inputBuffer, MAXINPUT, stdin); // read the user's input
    sscanf(inputBuffer, "%c", &runOn); // format and dump the user's input

    // Calculate the number of neuron/weight values we need space for and also the first Neuron/Weight index for each layer
    firstNeuronIndexPerLayer[0] = 0;  // input layer's first neuron starts at 0
    firstWeightIndexPerLayer[0] = -1; // input layer has no weights, put -1 just for fun
    numberOfWeightsPerLayer[0] = 0;   // input layer has no weights
    numberOfNeuronsTotal = numberOfNeuronsPerLayer[0]; // start by counting the neurons in input layer
    numberOfWeightsTotal = 0; // input layer has no weights
    for (int i = 1; i < numberOfLayers; i++) {
        firstNeuronIndexPerLayer[i] = numberOfNeuronsTotal;
        firstWeightIndexPerLayer[i] = numberOfWeightsTotal;
        numberOfWeightsPerLayer[i] = numberOfNeuronsPerLayer[i - 1] * numberOfNeuronsPerLayer[i];
        numberOfNeuronsTotal = numberOfNeuronsTotal + numberOfNeuronsPerLayer[i];
        numberOfWeightsTotal = numberOfWeightsTotal + (numberOfWeightsPerLayer[i]);
    }

    // dynamically allocate memory to store the neuron values, weight values, and outputExpected output values
    neurons = (double*) malloc(numberOfNeuronsTotal * sizeof(double)); //malloc allocates a chunk of host memory
    weights = (double*) malloc(numberOfWeightsTotal * sizeof(double)); //malloc allocates a chunk of host memory
    neuronErrors = (double*) malloc(numberOfNeuronsTotal * sizeof(double)); //malloc allocates a chunk of host memory
    outputExpected = (double*) malloc(numberOfNeuronsPerLayer[numberOfLayers - 1] * sizeof(double)); //malloc allocates a chunk of host memory

    // initialize every neuron and weight value to zero (clean up any garbage we may have picked up)
    printf("Starting init step now...\n");
    initArrayToZeros(neurons, numberOfNeuronsTotal);
    printf("\n");
    initArrayToRandomDoubles(weights, numberOfWeightsTotal);

    printf("initNeurons & initWeights successful!\n\n");

    if (runOn == 'd') {
        printf("Allocating GPU device memory and copying host values over...\n");

        // allocate device memory for device variables and copy host values to device copies
        hipMalloc((void **) &deviceNumberOfNeuronsPerLayer, numberOfLayers * sizeof(int)); //hipMalloc allocates a chunk of device memory
        hipMalloc((void **) &deviceNumberOfWeightsPerLayer, numberOfLayers * sizeof(int)); //hipMalloc allocates a chunk of device memory
        hipMalloc((void **) &deviceNeurons, (numberOfNeuronsTotal * sizeof(double))); //hipMalloc allocates a chunk of device memory
        hipMalloc((void **) &deviceWeights, (numberOfWeightsTotal * sizeof(double))); //hipMalloc allocates a chunk of device memory
        hipMalloc((void **) &deviceWeightCosts, (numberOfWeightsTotal * sizeof(double))); //hipMalloc allocates a chunk of device memory
        hipMemcpy(deviceNumberOfNeuronsPerLayer, numberOfNeuronsPerLayer, (numberOfLayers * sizeof(int)), hipMemcpyHostToDevice); //hipMemcpy copies host values to device copies
        hipMemcpy(deviceNumberOfWeightsPerLayer, numberOfWeightsPerLayer, (numberOfLayers * sizeof(int)), hipMemcpyHostToDevice); //hipMemcpy copies host values to device copies
        hipMemcpy(deviceNeurons, neurons, (numberOfNeuronsTotal * sizeof(double)), hipMemcpyHostToDevice); //hipMemcpy copies host values to device copies
        hipMemcpy(deviceWeights, weights, (numberOfWeightsTotal * sizeof(double)), hipMemcpyHostToDevice); //hipMemcpy copies host values to device copies

        printf("Allocation successful!\n\n");
    }

    for (int i = 0; i < epochs; i++) {
        // HOST - LOADINPUT, FEEDFORWARD, & BACKPROPAGATE
        printf("Starting load input step now...\n");
        loadInput(neurons, numberOfNeuronsPerLayer[0]); // load some random input for feedforward testing
        printarray("neurons", neurons, numberOfNeuronsTotal);

        printf("Starting feedforward step now...\n");
        feedforwardWithHost(neurons, weights, numberOfLayers, numberOfNeuronsPerLayer, numberOfWeightsPerLayer, firstNeuronIndexPerLayer,
                firstWeightIndexPerLayer); // feed the input forward

        printf("Network state post feedforward:\n");
        printarray("neurons", neurons, numberOfNeuronsTotal);
        printarray("weights", weights, numberOfWeightsTotal);

        printf("Generating random training labels for testing backpropagation now...\n");
        loadInput(outputExpected, numberOfNeuronsPerLayer[numberOfLayers - 1]); // load some random input for backpropagation testing
        printarray("outputExpected", outputExpected, numberOfNeuronsPerLayer[numberOfLayers - 1]);

        printf("Starting backpropagation step now...\n");
        backpropagateWithHost(outputExpected, neurons, weights, neuronErrors, numberOfLayers, numberOfNeuronsPerLayer, numberOfWeightsPerLayer,
                firstNeuronIndexPerLayer, firstWeightIndexPerLayer, learningRate); // calculate and back propagate errors

        printf("Network state post backpropagation:\n");
        printarray("neurons", neurons, numberOfNeuronsTotal);
        printarray("weights", weights, numberOfWeightsTotal);
    }

    printf("Press enter to free dynamically allocated memory.\n~");
    fgets(inputBuffer, MAXINPUT, stdin); // read the user's input

    printf("Freeing dynamically allocated memory...");

    // free the chunks of host memory that were dynamically allocated by malloc
    free(numberOfNeuronsPerLayer);
    free(numberOfWeightsPerLayer);
    free(firstNeuronIndexPerLayer);
    free(firstWeightIndexPerLayer);
    free(neurons);
    free(weights);
    free(neuronErrors);
    free(outputExpected);

    if (runOn == 'd') {
        // free the chunks of device memory that were dynamically allocated by hipMalloc
        hipFree(deviceNumberOfNeuronsPerLayer);
        hipFree(deviceNumberOfWeightsPerLayer);
        hipFree(deviceNeurons);
        hipFree(deviceWeights);
        hipFree(deviceWeightCosts);
    }

    printf("Memory freed!\n");
*/
    printf("%s will now end. ", argv[0]);
    printf("Press enter to end.\n~");
    fgets(inputBuffer, MAXINPUT, stdin); // read the user's input
} //end main method
