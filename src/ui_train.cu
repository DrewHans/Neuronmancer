/*******************************************************************************************
 * Filename: ui_train.cu
 * Author: Drew Hans (github.com/drewhans555)
 * Description: This file contains the user interface for training a Neuronmancer model.
 *******************************************************************************************
 */

/* ui_train method - user interface for training a model */
void ui_train() {
    // declare variables needed to store the model information
    char inputBuffer[MAXINPUT]; // store the user's input (gets recycled a lot)
    int tempInt; // store temp int input from user (used for determining whether to run on host or GPU device)
    int numberOfLayers; // store the total number of layers in the network
    int numberOfNeuronsTotal; // store the total number of neurons in our neural network
    int numberOfWeightsTotal; // store the total number of weights in our neural network
    int* numberOfNeuronsPerLayer; // store the total number of neurons in each layer in our neural network in a 1d array of size numberOfLayers
    int* numberOfWeightsPerLayer; // store the total number of weights between each layer in our neural network in a 1d array of size numberOfLayers-1
    int* firstNeuronIndexPerLayer; // store the indexes of each layer's first neuron value
    int* firstWeightIndexPerLayer; // store the indexes of each layer's first weight value
    Activation* activationsPerLayer; // store the activation of each layer
    double* neurons; // store the neuron values of our neural network in a 1d array of size neuronSize (1d arrays are easy to work with in CUDA)
    double* weights; // store the weight values of our neural network in a 1d array of size weightSize (1d arrays are easy to work with in CUDA)
    double* biases; // store the biases values of our neural network in a 1d array of size weightSize (1d arrays are easy to work with in CUDA)
    double* outputExpected; // store the outputExpected output values for backpropagation
    double* neuronErrors; // store the error "cost" of each neuron during backpropagation
    int epochs; // store the number of epochs for training
    double learningRate; // store the rate that our network will learn

    int myPatience = 2; // stores the amount of patience I have for the user's nonsense

    // initialize pointers with malloc (will be resized in readmodel.cu later)
    numberOfNeuronsPerLayer = (int *) malloc(sizeof(int));
    if (numberOfNeuronsPerLayer == NULL) {
        onMallocError(sizeof(int));
    }

    numberOfWeightsPerLayer = (int *) malloc(sizeof(int));
    if (numberOfWeightsPerLayer == NULL) {
        onMallocError(sizeof(int));
    }

    firstNeuronIndexPerLayer = (int *) malloc(sizeof(int));
    if (firstNeuronIndexPerLayer == NULL) {
        onMallocError(sizeof(int));
    }

    firstWeightIndexPerLayer = (int *) malloc(sizeof(int));
    if (firstWeightIndexPerLayer == NULL) {
        onMallocError(sizeof(int));
    }

    activationsPerLayer = (Activation *) malloc(sizeof(Activation));
    if (activationsPerLayer == NULL) {
        onMallocError(sizeof(Activation));
    }

    weights = (double *) malloc(sizeof(double));
    if (weights == NULL) {
        onMallocError(sizeof(double));
    }

    biases = (double *) malloc(sizeof(double));
    if (biases == NULL) {
        onMallocError(sizeof(double));
    }

    printf("Lets train an artificial neural network!\n");
    printf("Searching ./nmModel for files...\n");

    readModel(&numberOfLayers, &numberOfNeuronsTotal, &numberOfWeightsTotal, numberOfNeuronsPerLayer, numberOfWeightsPerLayer, firstNeuronIndexPerLayer,
            firstWeightIndexPerLayer, weights, biases, &learningRate, &epochs);

    printf("...files found!\n");

#ifdef DEBUG
    // print out information read from disk
    printf("epochs                 = %d\n", epochs);
    printf("learningRate           = %lf\n", learningRate);
    printf("numberOfLayers         = %d\n", numberOfLayers);
    printf("numberOfNeuronsTotal   = %d\n", numberOfNeuronsTotal);// remember, numberOfNeuronsTotal equals numberOfBiasesTotal
    printf("numberOfWeightsTotal   = %d\n", numberOfWeightsTotal);

    for(int i = 0; i < numberOfLayers; i++) {
        printf("numberOfNeuronsPerLayer[%d]  = %d\n", i, numberOfNeuronsPerLayer[i]);
        printf("numberOfWeightsPerLayer[%d]  = %d\n", i, numberOfWeightsPerLayer[i]);
        printf("firstNeuronIndexPerLayer[%d] = %d\n", i, firstNeuronIndexPerLayer[i]);
        printf("firstWeightIndexPerLayer[%d] = %d\n", i, firstWeightIndexPerLayer[i]);
        if(activationsPerLayer[i] == SIGMACT) {
            printf("activationsPerLayer[%d] = SIGMOID\n", i);
        } else if(activationsPerLayer[i] == RELUACT) {
            printf("activationsPerLayer[%d] = RELU\n", i);
        } else if(activationsPerLayer[i] == TANHACT) {
            printf("activationsPerLayer[%d] = TANH\n", i);
        } else {
            printf("activationsPerLayer[%d] = %d\n", i, activationsPerLayer[i]);
        }
    }

    printarray("biases", biases, numberOfNeuronsTotal);
    printarray("weights", weights, numberOfWeightsTotal);
#endif

    // malloc memory for uninitialized arrays
    neurons = (double *) malloc(numberOfNeuronsTotal * sizeof(double));
    if (neurons == NULL) {
        onMallocError(numberOfNeuronsTotal * sizeof(double));
    }

    outputExpected = (double *) malloc(numberOfNeuronsPerLayer[numberOfLayers - 1] * sizeof(double));
    if (outputExpected == NULL) {
        onMallocError(numberOfNeuronsPerLayer[numberOfLayers - 1] * sizeof(double));
    }

    neuronErrors = (double *) malloc(numberOfNeuronsTotal * sizeof(double));
    if (neuronErrors == NULL) {
        onMallocError(numberOfNeuronsTotal * sizeof(double));
    }

    // get user input for running on CPU or GPU
    tempInt = 'z'; // assign 'z' to enter loop
    while (tempInt != 'h' || tempInt != 'H' || tempInt != 'd' || tempInt != 'D') {
        // get the activation for layer i
        printf("Do you want to train on the host machine or GPU device?\nEnter h for host or d for device:\n~");
        fgets(inputBuffer, MAXINPUT, stdin); // read the user's input
        sscanf(inputBuffer, "%d", &tempInt); // format and dump the user's input
        if (tempInt != 'h' || tempInt != 'H' || tempInt != 'd' || tempInt != 'D') {
            onInvalidInput(myPatience);
            myPatience--;
        }
    }
    myPatience = 2; // restore my patience

    if (tempInt != 'h' || tempInt != 'H') {
        printf("Today we keep tradition, looks like we're training on the host machine!\n");

        // TODO: START HOST TRAINING
        printf("Beginning training now...");

        // TODO: do(LOADINPUT, FEEDFORWARD, COMPAREOUTPUT, BACKPROPAGATEERRS) for all samples in batch, WEIGHTUPDATE & BIASUPDATE, then repeat until i == epochs
        for (int i = 0; i < epochs; i++) {
            //printf("Starting load input step now...\n");
            loadInput(neurons, numberOfNeuronsPerLayer[0]); // load some random input for feedforward testing
            //printarray("neurons", neurons, numberOfNeuronsTotal);

            //printf("Starting feedforward step now...\n");
            // feed the input forward
            feedforwardWithHost(neurons, weights, biases, numberOfLayers, numberOfNeuronsPerLayer, numberOfWeightsPerLayer, firstNeuronIndexPerLayer,
                    firstWeightIndexPerLayer);

            //printf("Network state post feedforward:\n");
            //printarray("neurons", neurons, numberOfNeuronsTotal);
            //printarray("weights", weights, numberOfWeightsTotal);

            //printf("Generating random training labels for testing backpropagation now...\n");
            loadInput(outputExpected, numberOfNeuronsPerLayer[numberOfLayers - 1]); // load some random input for backpropagation testing
            //printarray("outputExpected", outputExpected, numberOfNeuronsPerLayer[numberOfLayers - 1]);

            //printf("Starting backpropagation step now...\n");
            // calculate and back propagate errors
            backpropagateWithHost(outputExpected, neurons, weights, biases, neuronErrors, numberOfLayers, numberOfNeuronsPerLayer, numberOfWeightsPerLayer,
                    firstNeuronIndexPerLayer, firstWeightIndexPerLayer);

            // use error signal (neuronErrors) to update the weights and biases
            updateWeights(neurons, weights, neuronErrors, numberOfLayers, numberOfNeuronsPerLayer, firstNeuronIndexPerLayer, firstWeightIndexPerLayer,
                    learningRate);
            updateBiases(neurons, biases, neuronErrors, numberOfNeuronsTotal, learningRate);

            //printf("Network state post backpropagation:\n");
            //printarray("neurons", neurons, numberOfNeuronsTotal);
            //printarray("weights", weights, numberOfWeightsTotal);
            if (i % 10 == 0) {
                printf("...%d epochs complete...", i);
            }

        }

    } else if (tempInt != 'd' || tempInt != 'D') {
        printf("Today we break with tradition, looks like we're training on the GPU device!\n");
        // declare our device variables
        double* devNeurons;
        double* devWeights;
        double* devBiases;
        double* devNeuronErrors;

        // declare our cudaStatus variable
        hipError_t cudaStatus;

        // run on GPU 0, change this on a multi-GPU system.
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            onFailToSetGPUDevice();
        }

        printf("Allocating GPU device memory and copying host values over...\n");

        // allocate device memory for device variables and copy host values to device copies
        cudaStatus = hipMalloc((void **) &devNeurons, (numberOfNeuronsTotal * sizeof(double))); //hipMalloc allocates a chunk of device memory
        if (cudaStatus != hipSuccess) {
            onCudaMallocError(numberOfNeuronsTotal * sizeof(double));
        }

        cudaStatus = hipMalloc((void **) &devWeights, (numberOfWeightsTotal * sizeof(double))); //hipMalloc allocates a chunk of device memory
        if (cudaStatus != hipSuccess) {
            onCudaMallocError(numberOfWeightsTotal * sizeof(double));
        }

        cudaStatus = hipMalloc((void **) &devBiases, (numberOfNeuronsTotal * sizeof(double))); //hipMalloc allocates a chunk of device memory
        if (cudaStatus != hipSuccess) {
            onCudaMallocError(numberOfNeuronsTotal * sizeof(double));
        }

        cudaStatus = hipMalloc((void **) &devNeuronErrors, (numberOfNeuronsTotal * sizeof(double))); //hipMalloc allocates a chunk of device memory
        if (cudaStatus != hipSuccess) {
            onCudaMallocError(numberOfNeuronsTotal * sizeof(double));
        }

        cudaStatus = hipMemcpy(devNeurons, neurons, (numberOfNeuronsTotal * sizeof(double)), hipMemcpyHostToDevice); //hipMemcpy copies host values to device copies
        if (cudaStatus != hipSuccess) {
            onCudaMemcpyError("numberOfNeuronsTotal");
        }

        cudaStatus = hipMemcpy(devWeights, weights, (numberOfWeightsTotal * sizeof(double)), hipMemcpyHostToDevice); //hipMemcpy copies host values to device copies
        if (cudaStatus != hipSuccess) {
            onCudaMemcpyError("numberOfWeightsTotal");
        }
        printf("...allocation successful!\n");

        // TODO: START GPU DEVICE TRAINING
        // use getDeviceProperties helper function to determine the numBlocks and threadsPerBlock before launching CUDA Kernels
        int numBlocks = 5; // set 5 as default, should be equal to the number of SMs on the GPU device
        int threadsPerBlock = 32; // set 32 as default, should be equal to the warpsize on the GPU device
        getDeviceProperties(&numBlocks, &threadsPerBlock);

        // TODO: do(LOADINPUT, FEEDFORWARD, COMPAREOUTPUT, BACKPROPAGATEERRS) for all samples in batch, WEIGHTUPDATE & BIASUPDATE, then repeat until i == epochs

        // for each node in the output layer, calculate the output error (spawn 1 thread for each neuron in the output layer)
        int outputLayerIndex = numberOfLayers - 1;

        // for each layer l between output and input, visit in reverse order, backpropagate error values and update weights
        for (int l = outputLayerIndex - 1; l > 0; l--) {
            // for each node in layer l, use error signal (devNeuronErrors) to update the devWeights and devBiases
            // spawn 1 block for each neuron in layer l and, in each block, spawn 1 thread for each neuron in layer l+1
            weightUpdateKernel<<<numberOfNeuronsPerLayer[l], numberOfNeuronsPerLayer[l + 1]>>>(devNeurons, devWeights, devNeuronErrors,
                    numberOfNeuronsPerLayer[l], numberOfNeuronsPerLayer[l + 1], numberOfWeightsPerLayer[l + 1], firstNeuronIndexPerLayer[l],
                    firstNeuronIndexPerLayer[l + 1], learningRate);
            hipDeviceSynchronize(); // tell host to wait for device to finish previous kernel
        }
        biasUpdateKernel<<<numBlocks, threadsPerBlock>>>(devNeurons, devBiases, devNeuronErrors, numberOfNeuronsTotal, learningRate);
        hipDeviceSynchronize(); // tell host to wait for device to finish previous kernel

        // TODO: COPY DEVICE VARIABLE VALUES BACK TO HOST

        printf("Press enter to free dynamically allocated GPU device memory.\n~");
        fgets(inputBuffer, MAXINPUT, stdin); // read the user's input

        printf("Freeing dynamically allocated GPU device memory...");

        // free the chunks of device memory that were dynamically allocated by hipMalloc
        hipFree(devNeurons);
        hipFree(devWeights);
        hipFree(devNeuronErrors);
    }

    // TODO: SAVE TRAINED WEIGHTS AND BIASES TO DISK

    printf("Press enter to free dynamically allocated host memory.\n~");
    fgets(inputBuffer, MAXINPUT, stdin); // read the user's input

    printf("Freeing dynamically allocated host memory...");

    // free the chunks of host memory that were dynamically allocated by malloc
    free(numberOfNeuronsPerLayer);
    free(numberOfWeightsPerLayer);
    free(firstNeuronIndexPerLayer);
    free(firstWeightIndexPerLayer);
    free(neurons);
    free(weights);
    free(neuronErrors);
    free(outputExpected);

    printf("Press enter to return to the main menu:\n~");
    fgets(inputBuffer, MAXINPUT, stdin); // read the user's input
    printf("\n");
} //end ui_train method
