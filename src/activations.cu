
#include <hip/hip_runtime.h>
/*******************************************************************************************
 * Filename: activations.cu
 * Author: Drew Hans (github.com/drewhans555)
 * Description: This file contains the host and device activation functions and cudaKernels
 *              for using a single activation function on every neuron in a single layer.
 *******************************************************************************************
 */

/*
 * sigmoidFunction - a Sigmoid activation function
 * __host__ decoration tells NVCC this function should run on CPU, and be callable from the CPU host
 * __device__ decoration tells NVCC this function should run on GPU, and be callable from the GPU device
 * @params: d - a double value
 * @returns: the Sigmoid of d
 */
__host__ __device__ double sigmoidFunction(double d) {
    return 1.0 / (1.0 + exp(-1.0 * d));
} //end sigmoid activation function

/*
 * sigmoidDerivative - a Sigmoid derivative function
 * __host__ decoration tells NVCC this function should run on CPU, and be callable from the CPU host
 * __device__ decoration tells NVCC this function should run on GPU, and be callable from the GPU device
 * @params: d - a double value
 * @returns: the Sigmoid derivative of d
 */
__host__ __device__ double sigmoidDerivative(double d) {
    return sigmoidFunction(d) * (1.0 - sigmoidFunction(d));
} //end sigmoid derivative function

/*
 * tanhFunction - a TanH activation function
 * __host__ decoration tells NVCC this function should run on CPU, and be callable from the CPU host
 * __device__ decoration tells NVCC this function should run on GPU, and be callable from the GPU device
 * @params: d - a double value
 * @returns: the TanH of d
 */
__host__ __device__ double tanhFunction(double d) {
    return (2.0 / (1.0 + exp(-2.0 * d))) - 1.0;
} //end tanh activation function

/*
 * tanhDerivative - a TanH derivative function
 * __host__ decoration tells NVCC this function should run on CPU, and be callable from the CPU host
 * __device__ decoration tells NVCC this function should run on GPU, and be callable from the GPU device
 * @params: d - a double value
 * @returns: the TanH derivative of d
 */
__host__ __device__ double tanhDerivative(double d) {
    return 1.0 - pow(tanhFunction(d), 2.0);
} //end tanh derivative function

/*
 * reluFunction - a ReLU activation function
 * __host__ decoration tells NVCC this function should run on CPU, and be callable from the CPU host
 * __device__ decoration tells NVCC this function should run on GPU, and be callable from the GPU device
 * @params: d - a double value
 * @returns: the ReLU of d
 */
__host__ __device__ double reluFunction(double d) {
    if (d < 0) {
        return 0.0;
    } else {
        return d;
    }
} //end relu activation function

/*
 * reluDerivative - a ReLU derivative function
 * __host__ decoration tells NVCC this function should run on CPU, and be callable from the CPU host
 * __device__ decoration tells NVCC this function should run on GPU, and be callable from the GPU device
 * @params: d - a double value
 * @returns: the ReLU derivative of d
 */
__host__ __device__ double reluDerivative(double d) {
    if (d < 0) {
        return 0.0;
    } else {
        return 1.0;
    }
} //end relu derivative function

/*
 * cudaKernel_ActivateLayerUsingSigmoid - applies Sigmoid activation to every neuron in a layer
 * __global__ decoration tells NVCC this function should run on GPU, and be callable from the CPU host
 * @params: devNeurons - a pointer to an array of double values in GPU device memory
 * @params: indexOfFirstNeuronInLayer - the index of the first neuron in the layer
 * @params: numberOfNeuronsInLayer - the total number of neurons in the layer
 */
__global__ void cudaKernel_ActivateLayerUsingSigmoid(double* devNeurons, int indexOfFirstNeuronInLayer, int numberOfNeuronsInLayer) {
    volatile unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

    // check that this thread is within our desired range (extra threads may have been launched for GPU optimization)
    if (id < numberOfNeuronsInLayer) {
        devNeurons[indexOfFirstNeuronInLayer + id] = sigmoidFunction(devNeurons[indexOfFirstNeuronInLayer + id]);
    }
} //end cudaKernel_ActivateLayerUsingSigmoid function

/*
 * cudaKernel_ActivateLayerUsingTanh - applies TanH activation to every neuron in a layer
 * __global__ decoration tells NVCC this function should run on GPU, and be callable from the CPU host
 * @params: devNeurons - a pointer to an array of double values in GPU device memory
 * @params: indexOfFirstNeuronInLayer - the index of the first neuron in the layer
 * @params: numberOfNeuronsInLayer - the total number of neurons in the layer
 */
__global__ void cudaKernel_ActivateLayerUsingTanh(double* devNeurons, int indexOfFirstNeuronInLayer, int numberOfNeuronsInLayer) {
    volatile unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

    // check that this thread is within our desired range (extra threads may have been launched for GPU optimization)
    if (id < numberOfNeuronsInLayer) {
        devNeurons[indexOfFirstNeuronInLayer + id] = tanhFunction(devNeurons[indexOfFirstNeuronInLayer + id]);
    }
} //end cudaKernel_ActivateLayerUsingTanh function

/*
 * cudaKernel_ActivateLayerUsingRelu - applies ReLU activation to every neuron in a layer
 * __global__ decoration tells NVCC this function should run on GPU, and be callable from the CPU host
 * @params: devNeurons - a pointer to an array of double values in GPU device memory
 * @params: indexOfFirstNeuronInLayer - the index of the first neuron in the layer
 * @params: numberOfNeuronsInLayer - the total number of neurons in the layer
 */
__global__ void cudaKernel_ActivateLayerUsingRelu(double* devNeurons, int indexOfFirstNeuronInLayer, int numberOfNeuronsInLayer) {
    volatile unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

    // check that this thread is within our desired range (extra threads may have been launched for GPU optimization)
    if (id < numberOfNeuronsInLayer) {
        devNeurons[indexOfFirstNeuronInLayer + id] = reluFunction(devNeurons[indexOfFirstNeuronInLayer + id]);
    }
} //end cudaKernel_ActivateLayerUsingRelu function

