/*******************************************************************************************
 * Filename: functions_misc.cu
 * Author: Drew Hans (github.com/drewhans555)
 * Description: This file contains helper functions - simple array operations, dealing with
 *              critical errors, printing insults, increasing/decreasing thread size, etc.
 *******************************************************************************************
 */

/*
 * getDeviceProperties
 * @params: multiProcessorCount - a pointer an int value (stores multiProcessorCount of the device)
 * @params: warpSize - a pointer an int value (stores the warpSize of the device)
 */
void getDeviceProperties(int* multiProcessorCount, int* warpSize) {
    hipDeviceProp_t devProp; //initialize hipDeviceProp_t struct
    hipGetDeviceProperties(&devProp, 0); //getDeviceProperties of device 0 and stuff them into address of devProp

    /*
     //basic device information
     printf("Name:                                   %s\n", devProp.name);
     printf("Major revision number:                  %d\n", devProp.major);
     printf("Minor revision number:                  %d\n", devProp.minor);

     //grid, block, thread info
     printf("Clock rate:                             %d kHz\n", devProp.clockRate);
     printf("Number of multiprocessors:              %d multiprocessors\n", devProp.multiProcessorCount);
     printf("Warp size:                              %d threads\n", devProp.warpSize);
     printf("Maximum threads per block:              %d threads\n", devProp.maxThreadsPerBlock);
     for (int i = 0; i < 3; ++i)
     printf("Maximum dimension %d of block:          %d\n", i, devProp.maxThreadsDim[i]);
     for (int i = 0; i < 3; ++i)
     printf("Maximum dimension %d of grid:           %d\n", i, devProp.maxGridSize[i]);

     //memory info
     printf("Total registers per multiprocessor:     %d 32-bits each\n", devProp.regsPerMultiprocessor);
     printf("Total registers per block:              %d 32-bits each\n", devProp.regsPerBlock);
     printf("Total shared memory per block:          %lu bytes\n", devProp.sharedMemPerBlock);
     printf("Total global memory:                    %lu bytes\n", devProp.totalGlobalMem);

     printf("Maximum memory pitch:                   %lu\n", devProp.memPitch);
     printf("Total constant memory:                  %lu bytes\n", devProp.totalConstMem);

     //other info
     printf("Texture alignment:                      %lu\n", devProp.textureAlignment);
     printf("Concurrent copy and execution:          %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
     printf("Kernel execution timeout:               %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
     */

    *multiProcessorCount = devProp.multiProcessorCount;
    *warpSize = devProp.warpSize;
} //end getDeviceProperties function

/*
 * getOptimalThreadSize - finds the "optimal" number of threads 
 * @params: blocks - the int number of blocks we plan to launch on GPU
 * @params: threads - the int number of threads we have set to launch on GPU
 * @params: minimumThreadsNeeded - the int minimum number of threads needed
 * @params: gpuWarpsize - the int warpsize of the GPU
 * @returns: the int number of "optimal" threads to launch
 */
int getOptimalThreadSize(int blocks, int threads, int minimumThreadsNeeded, int gpuWarpsize) {
    // double or devide the number of threads until we have a number close to the number of neurons in right-layer
    if ((blocks*threads) < minimumThreadsNeeded) {
        while((blocks*threads) < minimumThreadsNeeded) {
            threads = threads * 2;
        }
    } else if ((threads > gpuWarpsize) && ((blocks*(threads/2)) > minimumThreadsNeeded)) {
        while((threads > gpuWarpsize) && ((blocks*(threads/2)) > minimumThreadsNeeded)) {
            threads = threads / 2;
        }
    }
    return threads;
}//end getOptimalThreadSize function

/*
 * initArrayToRandomDoubles
 * @params: a - a pointer to an array of double values
 * @params: n - the size of array a
 */
void initArrayToRandomDoubles(double** a, int n) {
    // generate random doubles in range [0, 1)
    for (int i = 0; i < n; i++) {
        srand (time(NULL)); // seed pseudo-random number generator with current time
(        *a)[i] = ((double) rand()) / ((double) RAND_MAX);
    }
} //end initArrayToRandomDoubles function

/*
 * initArrayToZeros
 * @params: a - a pointer to an array of double values
 * @params: n - the size of array a
 */
void initArrayToZeros(double** a, int n) {
    // set all neuron values to zero
    for (int i = 0; i < n; i++) {
        (*a)[i] = 0;
    }
} //end initArrayToZeros function

/*
 * printarray_double - prints out array double values to terminal
 * @params: name - a pointer to a char string
 * @params: a - a pointer to an array of double values
 * @params: n - the size of array a
 */
void printarray_double(const char* name, double* a, int n) {
    for (int i = 0; i < n; i++) {
        printf("%s[%d]=%lf\n", name, i, a[i]);
    }
    printf("\n");
} //end printarray function

/*
 * printarray_int - prints out array int values to terminal
 * @params: name - a pointer to a char string
 * @params: a - a pointer to an array of int values
 * @params: n - the size of array a
 */
void printarray_int(const char* name, int* a, int n) {
    for (int i = 0; i < n; i++) {
        printf("%s[%d]=%d\n", name, i, a[i]);
    }
    printf("\n");
} //end printarray function

/*
 * printFarewellMSG - prints out one final insult before we crash
 */
void printFarewellMSG() {
    printf("Sorry, I did everything I could but it looks like I'm crashing...\n...\n...your computer sucks, good-bye.\n");
} //end printFarewellMSG function

void onCudaKernelLaunchFailure(char* kernel, hipError_t cudaStatus) {
    printf("%s launch failed: %s\n", kernel, hipGetErrorString(cudaStatus));
    printFarewellMSG();
    exit(1);
}

void onCudaDeviceSynchronizeError(char* kernel, hipError_t cudaStatus) {
    printf("hipDeviceSynchronize returned error code %d after launching %s!\n", cudaStatus, kernel);
    printFarewellMSG();
    exit(1);
}

/*
 * onCudaMallocError - SOS, we're going down
 * @params: size - the size of the device memory that we couldn't allocate
 */
void onCudaMallocError(int size) {
    printf("ERROR: Failed to hipMalloc %d of memory!\n", size);
    printFarewellMSG();
    exit(1);
} //end onCudaMallocError function

/*
 * onCudaMemcpyError - SOS, we're going down
 * @params: size - the name of the host variable that we couldn't copy
 */
void onCudaMemcpyError(const char* hostVariable) {
    printf("ERROR: Failed to hipMemcpy %s to device!\n", hostVariable);
    printFarewellMSG();
    exit(1);
} //end onCudaMemcpyError function

/*
 * onFailToSetGPUDevice - SOS, we're going down
 */
void onFailToSetGPUDevice() {
    printf("ERROR: Failed find GPU device!\n");
    printFarewellMSG();
    exit(1);
} //end onFailToSetGPUDevice function

/*
 * onFileOpenError - SOS, we're going down
 * @params: path - file that failed to open
 */
void onFileOpenError(const char* path) {
    printf("ERROR: Failed to open %s!\n", path);
    printFarewellMSG();
    exit(1);
} //end onFileOpenError function

/*
 * onFileReadError - SOS, we're going down
 * @params: path - file that failed to read
 */
void onFileReadError(const char* path) {
    printf("ERROR: Failed to read value from file %s!\n", path);
    printFarewellMSG();
    exit(1);
} //end onFileReadError function

/*
 * onInvalidInput - prints out insults when the user screws up (silly humans)
 * @params: myPatience - the current state of my patience, represented as an int
 */
void onInvalidInput(int myPatience) {
    if (myPatience == 2) {
        printf("Looks like you entered an illegal value... you're testing my patience, try again!\n\n");
    } else if (myPatience == 1) {
        printf("That's the second time you've entered an illegal value... do you think this is funny? Try again!\n\n");
    } else if (myPatience == 0) {
        printf("Sigh... you just can't do anything right, can you?\n\n");
    } else {
        printf("Look dude, I've got all day. If you wanna keep wasting your time then that's fine by me. You know what you're supposed to do.\n\n");
    }
} //end onInvalidInput function

/*
 * onMallocError - SOS, we're going down
 * @params: size - the size of the memory that we couldn't allocate
 */
void onMallocError(int size) {
    printf("ERROR: Failed to malloc %d of memory!\n", size);
    printFarewellMSG();
    exit(1);
} //end onMallocError function
