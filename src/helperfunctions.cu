
#include <hip/hip_runtime.h>
/*******************************************************************************************
 * Filename: helperfunctions.cu
 * Author: Drew Hans (github.com/drewhans555)
 * Description: This file contains helper functions - simple array operations, dealing with
 *              critical errors, printing insults, etc.
 *******************************************************************************************
 */

/*
 * getDeviceProperties
 * @params: multiProcessorCount - a pointer an int value (stores multiProcessorCount of the device)
 * @params: warpSize - a pointer an int value (stores the warpSize of the device)
 */
void getDeviceProperties(int* multiProcessorCount, int* warpSize) {
    hipDeviceProp_t devProp; //initialize cudaDeviceProp struct
    hipGetDeviceProperties(&devProp, 0); //getDeviceProperties of device 0 and stuff them into address of devProp

    /*
     //basic device information
     printf("Name:                                   %s\n", devProp.name);
     printf("Major revision number:                  %d\n", devProp.major);
     printf("Minor revision number:                  %d\n", devProp.minor);

     //grid, block, thread info
     printf("Clock rate:                             %d kHz\n", devProp.clockRate);
     printf("Number of multiprocessors:              %d multiprocessors\n", devProp.multiProcessorCount);
     printf("Warp size:                              %d threads\n", devProp.warpSize);
     printf("Maximum threads per block:              %d threads\n", devProp.maxThreadsPerBlock);
     for (int i = 0; i < 3; ++i)
     printf("Maximum dimension %d of block:          %d\n", i, devProp.maxThreadsDim[i]);
     for (int i = 0; i < 3; ++i)
     printf("Maximum dimension %d of grid:           %d\n", i, devProp.maxGridSize[i]);

     //memory info
     printf("Total registers per multiprocessor:     %d 32-bits each\n", devProp.regsPerMultiprocessor);
     printf("Total registers per block:              %d 32-bits each\n", devProp.regsPerBlock);
     printf("Total shared memory per block:          %lu bytes\n", devProp.sharedMemPerBlock);
     printf("Total global memory:                    %lu bytes\n", devProp.totalGlobalMem);

     printf("Maximum memory pitch:                   %lu\n", devProp.memPitch);
     printf("Total constant memory:                  %lu bytes\n", devProp.totalConstMem);

     //other info
     printf("Texture alignment:                      %lu\n", devProp.textureAlignment);
     printf("Concurrent copy and execution:          %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
     printf("Kernel execution timeout:               %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
     */

    *multiProcessorCount = devProp.multiProcessorCount;
    *warpSize = devProp.warpSize;
} //end getDeviceProperties

/*
 * initArrayToRandomDoubles
 * @params: a - a pointer to an array of double values
 * @params: n - the size of array a
 */
void initArrayToRandomDoubles(double** a, int n) {
    // generate random doubles in range [0, 1)
    for (int i = 0; i < n; i++) {
        (*a)[i] = ((double) rand()) / ((double) RAND_MAX);
    }
} //end initArrayToRandomDoubles method

/*
 * initArrayToZeros
 * @params: a - a pointer to an array of double values
 * @params: n - the size of array a
 */
void initArrayToZeros(double** a, int n) {
    // set all neuron values to zero
    for (int i = 0; i < n; i++) {
        (*a)[i] = 0;
    }
} //end initArrayToZeros method

/*
 * printarray - prints out array values to terminal
 * @params: name - a pointer to a char string
 * @params: a - a pointer to an array of double values
 * @params: n - the size of array a
 */
void printarray(const char* name, double* a, int n) {
    for (int i = 0; i < n; i++) {
        printf("%s[%d]=%lf\n", name, i, a[i]);
    }
    printf("\n");
} //end printarray method

/*
 * printFarewellMSG - prints out one final insult before we crash
 */
void printFarewellMSG() {
    printf("Sorry, I did everything I could but it looks like I'm crashing...\n...\n...your computer sucks, good-bye.\n");
} //end printFarewellMSG method

/*
 * onCudaMallocError - SOS, we're going down
 * @params: size - the size of the device memory that we couldn't allocate
 */
void onCudaMallocError(int size) {
    printf("ERROR: Failed to hipMalloc %d of memory!\n", size);
    printFarewellMSG();
    exit(1);
} //end onCudaMallocError method

/*
 * onCudaMemcpyError - SOS, we're going down
 * @params: size - the name of the host variable that we couldn't copy
 */
void onCudaMemcpyError(const char* hostVariable) {
    printf("ERROR: Failed to hipMemcpy %s to device!\n", hostVariable);
    printFarewellMSG();
    exit(1);
} //end onCudaMemcpyError method

/*
 * onFileOpenError - SOS, we're going down
 * @params: path - file that failed to open
 */
void onFileOpenError(const char* path) {
    printf("ERROR: Failed to open %s!\n", path);
    printFarewellMSG();
    exit(1);
} //end onFileOpenError method

/*
 * onFileReadError - SOS, we're going down
 * @params: path - file that failed to read
 */
void onFileReadError(const char* path) {
    printf("ERROR: Failed to read value from file %s!\n", path);
    printFarewellMSG();
    exit(1);
} //end onFileReadError method

/*
 * onFailToSetGPUDevice - SOS, we're going down
 */
void onFailToSetGPUDevice() {
    printf("ERROR: Failed find GPU device!\n");
    printFarewellMSG();
    exit(1);
} //end onFailToSetGPUDevice

/*
 * onInvalidInput - prints out insults when the user screws up (silly humans)
 * @params: myPatience - the current state of my patience, represented as an int
 */
void onInvalidInput(int myPatience) {
    if (myPatience == 2) {
        printf("Looks like you entered an illegal value... you're testing my patience, try again!\n\n");
    } else if (myPatience == 1) {
        printf("That's the second time you've entered an illegal value... do you think this is funny? Try again!\n\n");
    } else if (myPatience == 0) {
        printf("Sigh... you just can't do anything right, can you?\n\n");
    } else {
        printf("Look dude, I've got all day. If you wanna keep wasting your time then that's fine by me. You know what you're supposed to do.\n\n");
    }
} //end onInvalidInput method

/*
 * onMallocError - SOS, we're going down
 * @params: size - the size of the memory that we couldn't allocate
 */
void onMallocError(int size) {
    printf("ERROR: Failed to malloc %d of memory!\n", size);
    printFarewellMSG();
    exit(1);
} //end onMallocError method
