
#include <hip/hip_runtime.h>
/*******************************************************************************************
 * Filename: helperfunctions.cu
 * Author: Drew Hans (github.com/drewhans555)
 * Description: This file contains helper functions - simple array operations, dealing 
                with critical errors, printing insults, etc.
 *******************************************************************************************
 */
 
/*
 * initArrayToRandomDoubles method
 * @params: a - a pointer to an array of double values
 * @params: n - the size of array a
 */
void initArrayToRandomDoubles(double* a, int n) {
    // generate random doubles in range [0, 1)
    for (int i = 0; i < n; i++) {
        a[i] = ((double) rand()) / ((double) RAND_MAX);
    }
}//end initArrayToRandomDoubles method

/*
 * initArrayToZeros method
 * @params: a - a pointer to an array of double values
 * @params: n - the size of array a
 */
void initArrayToZeros(double* a, int n) {
    // set all neuron values to zero
    for (int i = 0; i < n; i++) {
        a[i] = 0;
    }
}//end initArrayToZeros method

/*
 * printarray method - prints out array values to terminal
 * @params: name - a pointer to a char string
 * @params: array - a pointer to an array of double values
 * @params: n - the size of array
 */
void printarray(const char* name, double* array, int n) {
    for (int i = 0; i < n; i++) {
        printf("%s[%d]=%f\n", name, i, array[i]);
    }
    printf("\n");
}//end printarray method

/*
 * printFarewellMSG method - prints out one final insult
 */
void printFarewellMSG() {
    printf("Sorry, I did everything I could but it looks like I'm crashing...\n...\n...your computer sucks, good-bye.\n");
}//end printFarewellMSG method

/*
 * onFileOpenError method - SOS, we're going down
 * @params: path - file that failed to open
 */
void onFileOpenError(const char* path) {
    printf("ERROR: Failed to open file %s!\n", path);
    printFarewellMSG();
    exit(1);
}//end onFileOpenError method

/*
 * onInvalidInput method - prints out insults when the user screws up (silly humans)
 * @params: myPatience - the current state of my patience, represented as an int
 */
void onInvalidInput(int myPatience) {
    if(myPatience == 2) {
        printf("Looks like you entered an illegal value... you're testing my patience, try again!\n");
    } else if(myPatience == 1) {
        printf("That's the second time you've entered an illegal value... do you think this is funny? Try again!\n");
    } else if(myPatience == 0) {
        printf("Sigh... you just can't do anything right, can you?\n");
    } else {
        printf("Look dude, I've got all day. If you wanna keep wasting your time then that's fine by me. You know what you're supposed to do.\n");
    }
}//end onInvalidInput method

/*
 * onMallocError method - SOS, we're going down
 * @params: size - the size of the memory that we couldn't allocate
 */
void onMallocError(int size) {
    printf("ERROR: Failed to malloc %d of memory!\n", size);
    printFarewellMSG();
    exit(1);
}//end onMallocError method
