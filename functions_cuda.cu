#include "hip/hip_runtime.h"
/********************************************************************************
 * Filename: functions_cuda.cu
 * Author: Drew Hans (github.com/drewhans555)
 * Description: This file contains the functions needed to train networks on
 *              cuda-enabled GPU devices.
 ********************************************************************************
 */

#include "main.h"

/*
 * cuda_loadMNISTTrainingSetToDevice - loads the MNIST training set images and labels into constant memory
 */
void cuda_loadMNISTTrainingSetToDevice() {
    const size_t trainImagesSize = sizeof(MNIST_Image) * MNIST_TRAINING_SET_SIZE;
    const size_t trainLabelsSize = sizeof(MNIST_Label) * MNIST_TRAINING_SET_SIZE;

    // malloc host memory for MNIST training set data (used for copying values to device)
    MNIST_Image* trainImages = (MNIST_Image*) malloc(trainImagesSize);
    if (trainImages == NULL) {
        printf("Abort! Could not malloc memory to store trainImages!\n");
        exit(1);
    }

    MNIST_Label* trainLabels = (MNIST_Label*) malloc(trainLabelsSize);
    if (trainLabels == NULL) {
        printf("Abort! Could not malloc memory to store trainLabels!\n");
        exit(1);
    }

    // open MNIST files
    FILE* imageFile, *labelFile;
    imageFile = openMNISTImageFile(MNIST_TRAINING_SET_IMAGES_LOCATION);
    labelFile = openMNISTLabelFile(MNIST_TRAINING_SET_LABELS_LOCATION);

    // copy file data into host variables
    for (int sample = 0; sample < MNIST_TRAINING_SET_SIZE; sample++) {
        // read the next sample image and label
        trainImages[sample] = getImage(imageFile);
        trainLabels[sample] = getLabel(labelFile);
    }

    // declare cudaStatus variable to check for success of cuda operations
    hipError_t cudaStatus;

    // copy host memory values in trainImages into device global memory variable dev_trainImages
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(dev_trainImages), trainImages, trainImagesSize, 0, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not hipMemcpyToSymbol trainImages!\n");
        exit(1);
    }

    // copy host memory values in trainLabels into device constant memory variable dev_trainLabels
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(dev_trainLabels), trainLabels, trainLabelsSize, 0, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not hipMemcpyToSymbol trainLabels!\n");
        exit(1);
    }

    // free dynamically allocated host memory
    free(trainImages);
    free(trainLabels);

} //end cuda_loadMNISTTrainingSetToDevice function

/*
 * cuda_sigmoid - returns the Sigmoid activation of x
 * @params const float x - some floating-point value
 */
__device__ float cuda_sigmoid(const float x) {
    return (1.0 / (1.0 + exp((x * -1.0))));
} //end cuda_sigmoid device function

/*
 * sigmoidPrime - returns the Sigmoid derivative of x
 * @params const float x - some floating-point value
 */
__device__ float cuda_sigmoidPrime(const float x) {
    return cuda_sigmoid(x) * (1.0 - cuda_sigmoid(x));
} //end cuda_sigmoidPrime device function

/*
 * cuda_trainNetwork - performs a single feedforward, backpropagation, update weights, and update biases training cycle
 * @params InputLayer* il - pointer to an InputLayer struct on device
 * @params HiddenLayer* hl - pointer to a HiddenLayer struct on device
 * @params OutputLayer* ol - pointer to an OutputLayer struct on device
 * @params ExpectedOutput* expected - pointer to an ExpectedOutput struct on device
 * @params int sample - index of the sample we want to train on
 * @params unsigned int iBlocks - the "optimal" number of blocks for InputLayer cudakernels
 * @params unsigned int iThreads - the "optimal" number of threads for InputLayer cudakernels
 * @params unsigned int hBlocks - the "optimal" number of blocks for HiddenLayer cudakernels
 * @params unsigned int hThreads - the "optimal" number of threads for HiddenLayer cudakernels
 * @params unsigned int oBlocks - the "optimal" number of blocks for OutputLayer cudakernels
 * @params unsigned int oThreads - the "optimal" number of threads for OutputLayer cudakernels
 */
void cuda_trainNetwork(InputLayer* dev_il, HiddenLayer* dev_hl, OutputLayer* dev_ol, ExpectedOutput* dev_expected, int sample, unsigned int iBlocks,
        unsigned int iThreads, unsigned int hBlocks, unsigned int hThreads, unsigned int oBlocks, unsigned int oThreads) {

    // (A) Feedforward Step
    // (A1) FeedInputLayer
    cudakernel_feedInputLayer<<<iBlocks, iThreads>>>(dev_il, sample); // feed image pixel-values into input-layer

    // Check for any errors launching the kernel
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_feedInputLayer failed to launch!\n");
        exit(1);
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered after the launch
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_feedInputLayer failed to synchronize!\n");
        exit(1);
    }

    // (A2) FeedHiddenLayer
    cudakernel_feedHiddenLayer<<<hBlocks, hThreads>>>(dev_hl, dev_il);   // feed input-layer values into hidden-layer

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_feedHiddenLayer failed to launch!\n");
        exit(1);
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered after the launch
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_feedHiddenLayer failed to synchronize!\n");
        exit(1);
    }

    // (A3) FeedOutputLayer
    cudakernel_feedOutputLayer<<<oBlocks, oThreads>>>(dev_ol, dev_hl);   // feed hidden-layer values into output-layer

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_feedOutputLayer failed to launch!\n");
        exit(1);
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered after the launch
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_feedOutputLayer failed to synchronize!\n");
        exit(1);
    }

    // (B) Backpropagation Step
    cudakernel_getExpectedOutput<<<oBlocks, oThreads>>>(dev_expected, sample);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_getExpectedOutput failed to launch!\n");
        exit(1);
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered after the launch
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_getExpectedOutput failed to synchronize!\n");
        exit(1);
    }

    cudakernel_calculateOutputLayerDeltas<<<oBlocks, oThreads>>>(dev_ol, dev_expected);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_calculateOutputLayerDeltas failed to launch!\n");
        exit(1);
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered after the launch
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_calculateOutputLayerDeltas failed to synchronize!\n");
        exit(1);
    }

    cudakernel_calculateHiddenLayerDeltas<<<hBlocks, hThreads>>>(dev_hl, dev_ol);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_calculateHiddenLayerDeltas failed to launch!\n");
        exit(1);
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered after the launch
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_calculateHiddenLayerDeltas failed to synchronize!\n");
        exit(1);
    }

    // (C) Update Weights & Biases
    // (C1) update HiddenLayer's weights and biases
    cudakernel_updateHiddenLayerWeightsAndBiases<<<hBlocks, hThreads>>>(dev_hl, dev_il);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_updateHiddenLayerWeightsAndBiases failed to launch!\n");
        exit(1);
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered after the launch
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_updateHiddenLayerWeightsAndBiases failed to synchronize!\n");
        exit(1);
    }

    // (C2) update OutputLayer's weights and biases
    cudakernel_updateOutputLayerWeightsAndBiases<<<oBlocks, oThreads>>>(dev_ol, dev_hl);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_updateOutputLayerWeightsAndBiases failed to launch!\n");
        exit(1);
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered after the launch
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("Abort! cudakernel_updateOutputLayerWeightsAndBiases failed to synchronize!\n");
        exit(1);
    }

} //end cuda_trainNetwork function

/*
 * cuda_train - trains a neural network using a CUDA enabled GPU device
 * @params InputLayer* il - pointer to an InputLayer struct
 * @params HiddenLayer* hl - pointer to a HiddenLayer struct
 * @params OutputLayer* ol - pointer to an OutputLayer struct
 */
void cuda_train(InputLayer* il, HiddenLayer* hl, OutputLayer* ol) {

    // declare cudaStatus variable to check for success of cuda operations
    hipError_t cudaStatus;

    // run on GPU 0, this will need to be changed on a multi-GPU system
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not detect a CUDA-enabled GPU!\n");
        exit(1);
    }

    // load MNIST training set into constant memory on GPU device
    cuda_loadMNISTTrainingSetToDevice();

    // declare helper structs on the device
    InputLayer* dev_il;
    HiddenLayer* dev_hl;
    OutputLayer* dev_ol;
    ExpectedOutput* dev_expected;

    // allocate device memory for storing
    cudaStatus = hipMalloc((void **) &dev_il, sizeof(*dev_il));
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not hipMalloc device memory to store dev_il!\n");
        exit(1);
    }

    cudaStatus = hipMalloc((void **) &dev_hl, sizeof(*dev_hl));
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not hipMalloc device memory to store dev_hl!\n");
        exit(1);
    }

    cudaStatus = hipMalloc((void **) &dev_ol, sizeof(*dev_ol));
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not hipMalloc device memory to store dev_ol!\n");
        exit(1);
    }

    cudaStatus = hipMalloc((void **) &dev_expected, sizeof(*dev_expected));
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not hipMalloc device memory to store dev_expected!\n");
        exit(1);
    }

    // hipMemcpy host variable values into device copies
    cudaStatus = hipMemcpy(dev_il, il, sizeof(*il), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not hipMemcpy il to dev_il!\n");
        exit(1);
    }
    cudaStatus = hipMemcpy(dev_il->input, il->input, sizeof(uint8_t) * IL_SIZE, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not hipMemcpy il->input to dev_il->input!\n");
        exit(1);
    }

    cudaStatus = hipMemcpy(dev_hl, hl, sizeof(*hl), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not hipMemcpy hl to dev_hl!\n");
        exit(1);
    }
    cudaStatus = hipMemcpy(dev_hl->hNeuron, hl->hNeuron, sizeof(HLNeuron) * HL_SIZE, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not hipMemcpy hl->hNeuron to dev_hl->hNeuron!\n");
        exit(1);
    }

    cudaStatus = hipMemcpy(dev_ol, ol, sizeof(*ol), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not hipMemcpy ol to dev_ol!\n");
        exit(1);
    }
    cudaStatus = hipMemcpy(dev_ol->oNeuron, ol->oNeuron, sizeof(OLNeuron) * OL_SIZE, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not hipMemcpy ol->oNeuron to dev_ol->oNeuron!\n");
        exit(1);
    }

    /*printf("sizeof(InputLayer): %10d\n"
     "sizeof(HiddenLayer): %10d\n"
     "sizeof(OutputLayer): %10d\n"
     "sizeof(*il) =: %10d\n"
     "sizeof(*hl) =: %10d\n"
     "sizeof(*ol) =: %10d\n"
     "sizeof(ExpectedOutput) =: %10d\n", sizeof(InputLayer), sizeof(HiddenLayer), sizeof(OutputLayer), sizeof(*il), sizeof(*hl), sizeof(*ol),
     sizeof(ExpectedOutput));
     */

    // declare variables for holding the "optimal" number of blocks / threads for a given layer
    unsigned int iBlocks, iThreads; // "optimal" blocks / threads for input-layer cudakernels
    unsigned int hBlocks, hThreads; // "optimal" blocks / threads for hidden-layer cudakernels
    unsigned int oBlocks, oThreads; // "optimal" blocks / threads for output-layer cudakernels

    // calculate the "optimal" number of blocks / threads for each layer
    getOptimalBlocksAndThreads(&iBlocks, &iThreads, IL_SIZE);
    getOptimalBlocksAndThreads(&hBlocks, &hThreads, HL_SIZE);
    getOptimalBlocksAndThreads(&oBlocks, &oThreads, OL_SIZE);

    // begin training
    // for each epoch
    for (int epoch = 0; epoch < EPOCHS; epoch++) {

        // for each MNIST sample in the training set
        for (int sample = 0; sample < MNIST_TRAINING_SET_SIZE; sample++) {

            cuda_trainNetwork(dev_il, dev_hl, dev_ol, dev_expected, sample, iBlocks, iThreads, hBlocks, hThreads, oBlocks, oThreads);

            if (sample + 1 == 10000 || sample + 1 == 20000 || sample + 1 == 30000 || sample + 1 == 40000 || sample + 1 == 50000 || sample + 1 == 60000) {
                printf("   => sample %d of %d complete\n", sample + 1, MNIST_TRAINING_SET_SIZE);
            }

        }

        printf("--- epoch %d of %d complete ---\n", epoch + 1, EPOCHS);

    }

    // hipMemcpy device variable values into host variables
    cudaStatus = hipMemcpy(il, dev_il, sizeof(*dev_il), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not hipMemcpy dev_il to il!\n");
        exit(1);
    }

    cudaStatus = hipMemcpy(hl, dev_hl, sizeof(*dev_hl), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not hipMemcpy dev_hl to hl!\n");
        exit(1);
    }

    cudaStatus = hipMemcpy(ol, dev_ol, sizeof(*dev_ol), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        printf("Abort! Could not hipMemcpy dev_ol to ol!\n");
        exit(1);
    }

    // hipFree dynamically allocated device memory
    hipFree(dev_il);
    hipFree(dev_hl);
    hipFree(dev_ol);
    hipFree(dev_expected);

} //end cuda_train functions

/*
 * cudakernel_calculateHiddenLayerDeltas - calculates the delta values for output-layer neurons
 * __global__ decoration tells NVCC this function should run on GPU, and be callable from the CPU host
 * __restrict__ decoration tells NVCC this pointer will only be used to refer to the underlying data
 * @params HiddenLayer* dev_hl - pointer to an HiddenLayer struct on device
 * @params OutputLayer* dev_ol - pointer to an OutputLayer struct on device
 */
__global__ void cudakernel_calculateHiddenLayerDeltas(HiddenLayer* __restrict__ dev_hl, OutputLayer* __restrict__ dev_ol) {

    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x; // calculate the thread id

    // check that thread id is within our desired range (extra threads may have been launched for GPU optimization)
    if (id < HL_SIZE) {
        // declare a temporary variable in local memory for storing the delta
        float delta = 0.0; // helps us limit the number of reads/writes from/to global memory

        // for each oNeuron in OutputLayer
        for (int i = 0; i < OL_SIZE; i++) {
            // propagate ol->oNeuron[i]'s delta backwards
            delta += dev_ol->oNeuron[i].weight[id] * dev_ol->oNeuron[i].delta;
        }

        // calculate hl->hNeuron[i]'s delta
        dev_hl->hNeuron[id].delta = cuda_sigmoidPrime(dev_hl->hNeuron[id].weightedSum) * delta;
    }

} //end cudakernel_calculateHiddenLayerDeltas function

/*
 * cudakernel_calculateOutputLayerDeltas - calculates the delta values for output-layer neurons
 * __global__ decoration tells NVCC this function should run on GPU, and be callable from the CPU host
 * __restrict__ decoration tells NVCC this pointer will only be used to refer to the underlying data
 * @params OutputLayer* dev_il - pointer to an OutputLayer struct on device
 * @params ExpectedOutput* dev_expected - pointer to an ExpectedOutput struct on device
 */
__global__ void cudakernel_calculateOutputLayerDeltas(OutputLayer* __restrict__ dev_ol, ExpectedOutput* __restrict__ dev_expected) {

    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x; // calculate the thread id

    // check that thread id is within our desired range (extra threads may have been launched for GPU optimization)
    if (id < OL_SIZE) {
        dev_ol->oNeuron[id].delta = cuda_sigmoidPrime(dev_ol->oNeuron[id].weightedSum) * (dev_expected->value[id] - dev_ol->oNeuron[id].output);
    }

} //end cudakernel_calculateOutputLayerDeltas function

/*
 * cudakernel_feedInputLayer - feeds MNIST pixel values into the input-layer
 * __global__ decoration tells NVCC this function should run on GPU, and be callable from the CPU host
 * __restrict__ decoration tells NVCC this pointer will only be used to refer to the underlying data
 * @params InputLayer* dev_il - pointer to an InputLayer struct on device
 * @params MNIST_Image* dev_image - pointer to an MNIST_Image struct on device
 */
__global__ void cudakernel_feedInputLayer(InputLayer* dev_il, int sample) {

    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x; // calculate the thread id

    // check that thread id is within our desired range (extra threads may have been launched for GPU optimization)
    if (id < IL_SIZE) {
        // if dev_image->pixel[i] !0 then set dev_il->input[i] to 1, else set to 0
        dev_il->input[id] = (dev_trainImages[sample].pixel[id] ? 1 : 0);
    }

} //end cudakernel_feedInputLayer function

/*
 * cudakernel_feedHiddenLayer - feeds input-layer values into hidden-layer
 * __global__ decoration tells NVCC this function should run on GPU, and be callable from the CPU host
 * __restrict__ decoration tells NVCC this pointer will only be used to refer to the underlying data
 * @params HiddenLayer* dev_hl - pointer to a HiddenLayer struct on device
 * @params InputLayer* dev_il - pointer to an InputLayer struct on device
 */
__global__ void cudakernel_feedHiddenLayer(HiddenLayer* __restrict__ dev_hl, InputLayer* __restrict__ dev_il) {

    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x; // calculate the thread id

    // check that thread id is within our desired range (extra threads may have been launched for GPU optimization)
    if (id < HL_SIZE) {
        // declare a temporary variable in local memory for storing the weightedSum
        float weightedSum = 0.0; // helps us limit the number of reads/writes from/to global memory

        // for each input[i] to HLNeuron, add il->input[i] * hNeuron[id].weight[i] to HLNeuron's weighted sum
        for (int i = 0; i < IL_SIZE; i++) {
            weightedSum += dev_il->input[i] * dev_hl->hNeuron[id].weight[i];
        }

        // store weightedSum
        dev_hl->hNeuron[id].weightedSum = weightedSum;

        // apply sigmoid activation to hNeuron's weighted sum plus bias
        dev_hl->hNeuron[id].output = cuda_sigmoid(weightedSum + dev_hl->hNeuron[id].bias);
    }

} //end cudakernel_feedHiddenLayer function

/*
 * cudakernel_feedOutputLayer - feeds hidden-layer values into output-layer
 * __global__ decoration tells NVCC this function should run on GPU, and be callable from the CPU host
 * __restrict__ decoration tells NVCC this pointer will only be used to refer to the underlying data
 * @params OutputLayer* dev_ol - pointer to an OutputLayer struct on device
 * @params HiddenLayer* dev_hl - pointer to a HiddenLayer struct on device
 */
__global__ void cudakernel_feedOutputLayer(OutputLayer* __restrict__ dev_ol, HiddenLayer* __restrict__ dev_hl) {

    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x; // calculate the thread id

    // check that thread id is within our desired range (extra threads may have been launched for GPU optimization)
    if (id < OL_SIZE) {
        // declare a temporary variable in local memory for storing the weightedSum
        float weightedSum = 0.0; // helps us limit the number of reads/writes from/to global memory

        // for each input[i] to HLNeuron, add il->input[i] * hNeuron[id].weight[i] to HLNeuron's weighted sum
        for (int i = 0; i < OL_SIZE; i++) {
            weightedSum += dev_hl->hNeuron[i].output * dev_ol->oNeuron[id].weight[i];
        }

        // store weightedSum
        dev_ol->oNeuron[id].weightedSum = weightedSum;

        // apply sigmoid activation to the hln's weighted sum plus bias
        dev_ol->oNeuron[id].output = cuda_sigmoid(weightedSum + dev_ol->oNeuron[id].bias);
    }

} //end cudakernel_feedOutputLayer function

/*
 * cudakernel_getExpectedOutput - fills dev_expected with the expected output values
 * __global__ decoration tells NVCC this function should run on GPU, and be callable from the CPU host
 * __restrict__ decoration tells NVCC this pointer will only be used to refer to the underlying data
 * @params ExpectedOutput* dev_expected - pointer to an ExpectedOutput struct on device
 * @params int dev_mnistLabel - the label of an mnist sample on device
 */
__global__ void cudakernel_getExpectedOutput(ExpectedOutput* __restrict__ dev_expected, int sample) {

    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x; // calculate the thread id

    // check that thread id is within our desired range (extra threads may have been launched for GPU optimization)
    if (id < OL_SIZE) {
        // if id == dev_trainLabels[sample] set dev_expected->value[i] to 1, else set to 0
        dev_expected->value[id] = (id == dev_trainLabels[sample] ? 1 : 0);
    }

} //end cudakernel_getExpectedOutput function

/*
 * cudakernel_updateHiddenLayerWeightsAndBiases - updates the HiddenLayer hNeuron's weights and biases
 * __global__ decoration tells NVCC this function should run on GPU, and be callable from the CPU host
 * __restrict__ decoration tells NVCC this pointer will only be used to refer to the underlying data
 * @params HiddenLayer* dev_hl - pointer to a HiddenLayer struct on device
 * @params InputLayer* dev_il - pointer to an InputLayer struct on device
 */
__global__ void cudakernel_updateHiddenLayerWeightsAndBiases(HiddenLayer* __restrict__ dev_hl, InputLayer* __restrict__ dev_il) {

    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x; // calculate the thread id

    // check that thread id is within our desired range (extra threads may have been launched for GPU optimization)
    if (id < HL_SIZE) {
        // update each weight between InputLayer and hNeuron[id]
        for (int i = 0; i < IL_SIZE; i++) {
            dev_hl->hNeuron[id].weight[i] += LEARNING_RATE * dev_il->input[i] * dev_hl->hNeuron[id].delta;
        }

        // update hNeuron[id]'s bias
        dev_hl->hNeuron[id].bias += LEARNING_RATE * dev_hl->hNeuron[id].delta;
    }

} //end cudakernel_updateHiddenLayerWeightsAndBiases function

/*
 * cudakernel_updateOutputLayerWeightsAndBiases - updates the OutputLayer oNeuron's weights and biases
 * __global__ decoration tells NVCC this function should run on GPU, and be callable from the CPU host
 * __restrict__ decoration tells NVCC this pointer will only be used to refer to the underlying data
 * @params OutputLayer* dev_ol - pointer to an OutputLayer struct on device
 * @params HiddenLayer* dev_hl - pointer to a HiddenLayer struct on device
 */
__global__ void cudakernel_updateOutputLayerWeightsAndBiases(OutputLayer* __restrict__ dev_ol, HiddenLayer* __restrict__ dev_hl) {

    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x; // calculate the thread id

    // check that thread id is within our desired range (extra threads may have been launched for GPU optimization)
    if (id < OL_SIZE) {
        // update each weight between HiddenLayer and oNeuron[id]
        for (int i = 0; i < HL_SIZE; i++) {
            dev_ol->oNeuron[id].weight[i] += LEARNING_RATE * dev_hl->hNeuron[i].output * dev_ol->oNeuron[id].delta;
        }

        // update oNeuron[id]'s bias
        dev_ol->oNeuron[id].bias += LEARNING_RATE * dev_ol->oNeuron[id].delta;
    }

} //end cudakernel_updateOutputLayerWeightsAndBiases function

/*
 * getDeviceProperties - detects and stores the number of SMs and warpsize in arguments passed in
 * @params: multiProcessorCount - a pointer to an int value (stores multiProcessorCount of the device)
 * @params: warpSize - a pointer to an int value (stores the warpSize of the device)
 * @parmas: maxThreadsPerBlock - a pointer to an int value (stores the maxThreadsPerBlock of the device)
 */
void getDeviceProperties(unsigned int* multiProcessorCount, unsigned int* warpSize, unsigned int* maxThreadsPerBlock) {
    hipDeviceProp_t devProp; //initialize hipDeviceProp_t struct
    hipGetDeviceProperties(&devProp, 0); //getDeviceProperties of device 0 and stuff them into address of devProp

    /*
     //basic device information
     printf("Name:                                   %s\n", devProp.name);
     printf("Major revision number:                  %d\n", devProp.major);
     printf("Minor revision number:                  %d\n", devProp.minor);

     //grid, block, thread info
     printf("Clock rate:                             %d kHz\n", devProp.clockRate);
     printf("Number of multiprocessors:              %d multiprocessors\n", devProp.multiProcessorCount);
     printf("Warp size:                              %d threads\n", devProp.warpSize);
     printf("Maximum threads per block:              %d threads\n", devProp.maxThreadsPerBlock);
     for (int i = 0; i < 3; ++i)
     printf("Maximum dimension %d of block:          %d\n", i, devProp.maxThreadsDim[i]);
     for (int i = 0; i < 3; ++i)
     printf("Maximum dimension %d of grid:           %d\n", i, devProp.maxGridSize[i]);

     //memory info
     printf("Total registers per multiprocessor:     %d 32-bits each\n", devProp.regsPerMultiprocessor);
     printf("Total registers per block:              %d 32-bits each\n", devProp.regsPerBlock);
     printf("Total shared memory per block:          %lu bytes\n", devProp.sharedMemPerBlock);
     printf("Total global memory:                    %lu bytes\n", devProp.totalGlobalMem);

     printf("Maximum memory pitch:                   %lu\n", devProp.memPitch);
     printf("Total constant memory:                  %lu bytes\n", devProp.totalConstMem);

     //other info
     printf("Texture alignment:                      %lu\n", devProp.textureAlignment);
     printf("Concurrent copy and execution:          %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
     printf("Kernel execution timeout:               %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
     */

    *multiProcessorCount = devProp.multiProcessorCount;
    *warpSize = devProp.warpSize;
    *maxThreadsPerBlock = devProp.maxThreadsPerBlock;
} //end getDeviceProperties function

/*
 * getOptimalBlocksAndThread - finds the "optimal" number of blocks and threads needed to launch a kernel
 * @params: blocks - the int number of blocks we plan to launch on GPU
 * @params: threads - the int number of threads we have set to launch on GPU
 * @params: minimumThreadsNeeded - the int minimum number of threads needed (usually an array size)
 */
void getOptimalBlocksAndThreads(unsigned int* blocks, unsigned int* threads, const unsigned int minimumThreadsNeeded) {

    // get relevant information about GPU device
    unsigned int gpuMultiProcessorCount = 0;
    unsigned int gpuWarpSize = 0;
    unsigned int gpuMaxThreadsPerBlock = 0;
    getDeviceProperties(&gpuMultiProcessorCount, &gpuWarpSize, &gpuMaxThreadsPerBlock);

    // first 5 if statements should be sufficient for small kernel launches
    if (gpuWarpSize > minimumThreadsNeeded) {
        // set the minimum blocks and threads that we must have for any cudaKernel
        (*blocks) = 1;
        (*threads) = gpuWarpSize;
    } else if (gpuWarpSize * 2 > minimumThreadsNeeded && gpuWarpSize * 2 < gpuMaxThreadsPerBlock) {
        (*blocks) = 1;
        (*threads) = gpuWarpSize * 2;
    } else if (gpuWarpSize * 4 > minimumThreadsNeeded && gpuWarpSize * 4 < gpuMaxThreadsPerBlock) {
        (*blocks) = 1;
        (*threads) = gpuWarpSize * 4;
    } else if (gpuWarpSize * 8 > minimumThreadsNeeded && gpuWarpSize * 8 < gpuMaxThreadsPerBlock) {
        (*blocks) = 1;
        (*threads) = gpuWarpSize * 8;
    } else if (gpuWarpSize * 16 > minimumThreadsNeeded && gpuWarpSize * 16 < gpuMaxThreadsPerBlock) {
        // start adding blocks, probably easier than doubling the threads at this point
        (*blocks) = 1;
        (*threads) = gpuWarpSize * 16;
    } else if (gpuWarpSize * 16 > minimumThreadsNeeded && gpuWarpSize * 8 < gpuMaxThreadsPerBlock) {
        // start adding blocks, probably easier than doubling the threads at this point
        (*blocks) = 2;
        (*threads) = gpuWarpSize * 8;
    } else if (gpuWarpSize * 32 > minimumThreadsNeeded && gpuWarpSize * 8 < gpuMaxThreadsPerBlock) {
        // add another two blocks, still probably easier than doubling the threads
        (*blocks) = 4;
        (*threads) = gpuWarpSize * 8;
    } else if (gpuWarpSize * 64 > minimumThreadsNeeded && gpuWarpSize * 16 < gpuMaxThreadsPerBlock) {
        // okay, guess we're doubling threads then...
        (*blocks) = 4;
        (*threads) = gpuWarpSize * 16;
    } else if (gpuWarpSize * 128 > minimumThreadsNeeded && gpuWarpSize * 16 < gpuMaxThreadsPerBlock) {
        // okay, guess we're adding blocks and doubling threads...
        (*blocks) = 8;
        (*threads) = gpuWarpSize * 16;
    } else if (gpuWarpSize * 256 > minimumThreadsNeeded && gpuWarpSize * 16 < gpuMaxThreadsPerBlock) {
        // okay, guess we're adding blocks and doubling threads...
        (*blocks) = 16;
        (*threads) = gpuWarpSize * 16;
    } else if (gpuWarpSize * 512 > minimumThreadsNeeded && gpuWarpSize * 16 < gpuMaxThreadsPerBlock) {
        // this is getting ridiculous...
        (*blocks) = 32;
        (*threads) = gpuWarpSize * 16;
    } else if (gpuWarpSize * 1024 > minimumThreadsNeeded && gpuWarpSize * 16 < gpuMaxThreadsPerBlock) {
        // I hope this block of code never has to be executed... that poor GPU...
        (*blocks) = 64;
        (*threads) = gpuWarpSize * 16;
    } else {
        // heuristic time! now we brute force numbers (not even close to optimal, buy hey, we've got work to do)
        (*blocks) = minimumThreadsNeeded / (gpuWarpSize * 8);
        (*threads) = gpuMaxThreadsPerBlock / 2;
        while (((*blocks) * (*threads)) < minimumThreadsNeeded) {
            (*blocks) = (*blocks) + 1; // add another block
            if (((*blocks) * ((*threads) / 2)) > minimumThreadsNeeded) {
                // if adding another block allows us to cut our threads in half then do it.
                (*threads) = (*threads) / 2;
            }
        }
    }

} //end getOptimalThreadSize function
